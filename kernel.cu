#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <vector>
#include <random>
#include <cassert>

#include <cstdio>

constexpr int width = 10;
constexpr int height = 10;
constexpr int size = width * height;

constexpr int TILE_W = 14;
constexpr int TILE_H = 14;

bool cudaLogErrorInternal(hipError_t error, const char* file, uint32_t line, const char* function,
    const char* cmd) {
    if (error != hipSuccess) {
        fprintf(stderr, "Cuda Error %s:%u %s\n\t%s\nReturned error:\n\t%s:\t%s", file, line, function, cmd,
            hipGetErrorName(error), hipGetErrorString(error));
        return false;
    }
    return true;
}

#define cudaLogError(cmd) cudaLogErrorInternal((cmd), __FILE__, __LINE__, __PRETTY_FUNCTION__, #cmd)
#define cudaLogFatalError(cmd)                                                                                         \
  do {                                                                                                                 \
    if (!cudaLogErrorInternal((cmd), __FILE__, __LINE__, __PRETTY_FUNCTION__, #cmd))                                   \
      abort();                                                                                                         \
  } while (0)

__device__ int getIdx(int x, int y) {
    y += (y < 0) ? height : (y >= height) ? -height : 0;
    x += (x < 0) ? width : (x >= width) ? -width : 0;
    int i = (y ) * width + (x );
    return i;
}

__global__ void stepKernel(const uint8_t* state, uint8_t* out)
{
    int x = TILE_W * (blockIdx.x - 1) + (threadIdx.x - 1);
    int y = TILE_H * (blockIdx.y - 1) + (threadIdx.y - 1);

    __shared__ uint8_t tile[TILE_H + 2][TILE_W + 2];

    int C = getIdx(x, y);
    tile[threadIdx.y][threadIdx.x] = state[C];

    if (x >= width || y >= height || threadIdx.x == 0 || threadIdx.y == 0 || threadIdx.x > TILE_W || threadIdx.y > TILE_H) {
        return;
    }
    
    __syncthreads();

    x = threadIdx.x;
    y = threadIdx.y;
    int N = tile[y - 1][x];
    int NE = tile[y - 1][x + 1];
    int NW = tile[y - 1][x - 1];
    int E = tile[y][x + 1];
    int W = tile[y][x - 1];
    int S = tile[y + 1][x];
    int SE = tile[y + 1][x + 1];
    int SW = tile[y + 1][x - 1];

    int total = N + NE + NW + E + W + S + SE + SW;
    out[C] = tile[y][x] && total == 2 || total == 3;
}

hipError_t stepWithCuda(const std::vector<uint8_t>& a, std::vector<uint8_t>& b, uint8_t* dev_a, uint8_t* dev_b, hipStream_t stream)
{
    hipError_t cudaStatus;

    dim3 threadsPerBlock(TILE_W + 2, TILE_H + 2, 1);

    dim3 blocksPerGrid((width + TILE_W - 1) / TILE_W, (height + TILE_H - 1) / TILE_H, 1);
    stepKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (dev_a, dev_b);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }

    cudaStatus = hipMemcpyAsync(b.data(), dev_b, size * sizeof(uint8_t), hipMemcpyDeviceToHost, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpyAsync failed!");
        return cudaStatus;
    }

    return cudaStatus;
}

void draw(std::vector<uint8_t>& state) {
    for (int i = 0; i < width * height; ++i) {
        if (state[i]) { std::cout << "*"; }
        else { std::cout << "."; }

        if (i % width == width - 1) { std::cout << "\n"; }

    }
    for (int i = 0; i < 10; ++i) {
        std::cout << "\n";
    }
}

void app(std::vector<uint8_t>& state, std::vector<uint8_t>& out) {

    uint8_t* dev_a = 0; //should point to block of gpu memory
    uint8_t* dev_b = 0;


    hipStream_t stream{};
    cudaLogFatalError(hipStreamCreate(&stream));
    cudaLogFatalError(hipMalloc((void**)&dev_a, size * sizeof(uint8_t)));
    cudaLogFatalError(hipMalloc((void**)&dev_b, size * sizeof(uint8_t)));

    //async allows you to choose what stream, and returns immediately
    cudaLogFatalError(hipMemcpyAsync(dev_a, state.data(), size * sizeof(uint8_t), hipMemcpyHostToDevice, stream));

    cudaLogFatalError(stepWithCuda(state, out, dev_a, dev_b, stream));

    draw(state);
    while (1) {
        char key = 'x';
        // std::cin >> key;
        if (key == 'x')
        {
            stepWithCuda(state, out, dev_a, dev_b, stream);
            draw(out);
            std::swap(dev_a, dev_b);
        }
        else
        {
            break;
        }
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipStreamDestroy(stream); //"opaque type"
}

int main()
{
    std::vector<uint8_t> a;

    std::random_device rd;  // Will be used to obtain a seed for the random number engine
    std::mt19937 gen(rd()); // Standard mersenne_twister_engine seeded with rd()
    std::uniform_int_distribution<> distrib(false, true);

    for (int i = 0; i < width * height; ++i) {
        a.push_back(distrib(gen));
    }

    //a.resize(width * height);
    //a[0] = 1;
    //a[width] = 1;
    //a[width + 2] = 1;
    //a[width * 2] = 1;
    //a[width * 2 + 1] = 1;

    std::vector<uint8_t> b(a.size(), 0);
    app(a, b);
    cudaLogFatalError(hipDeviceReset());
    return 0;
}

